#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// cwtMultiAll.cu
//
// Code generation for function 'cwtMultiAll'
//

// Include files
#include "cwtMultiAll.h"
#include "cwtMultiAll_data.h"
#include "rt_nonfinite.h"
#include "MWCUFFTPlanManager.hpp"
#include "MWCudaDimUtility.hpp"
#include "hipfft/hipfft.h"
#include <cmath>

// Variable Definitions
static real_T psidft[19200];

static real_T cf[48];

static boolean_T gpuConstsCopied_cwtMultiAll;

// Function Declarations
static __global__ void cwtMultiAll_kernel1(const real_T FourierFactor,
                                           const real_T dv1[400],
                                           const real_T dv[48], real_T b_cf[48],
                                           real_T b_psidft[19200]);

static __global__ void cwtMultiAll_kernel10(creal_T cfsdft[19200]);

static __global__ void cwtMultiAll_kernel11(const creal_T cfsdft[19200],
                                            creal_T cfs[9600]);

static __global__ void cwtMultiAll_kernel12(const creal_T cfs[9600],
                                            const int32_T tIndex,
                                            creal_T cwtres[96000]);

static __global__ void cwtMultiAll_kernel2(const real_T data[2000],
                                           real_T xv[400]);

static __global__ void cwtMultiAll_kernel3(creal_T xdft[400]);

static __global__ void cwtMultiAll_kernel4(const creal_T xdft[400],
                                           creal_T cfsdft[19200],
                                           real_T b_psidft[19200]);

static __global__ void cwtMultiAll_kernel5(const real_T maxwavcf,
                                           const real_T FourierFactor,
                                           real_T coi[200]);

static __global__ void cwtMultiAll_kernel6(real_T b_cf[48], real_T f[48]);

static __global__ void cwtMultiAll_kernel7(const real_T data[2000],
                                           const int32_T tIndex,
                                           real_T xv[400]);

static __global__ void cwtMultiAll_kernel8(creal_T xdft[400]);

static __global__ void cwtMultiAll_kernel9(const creal_T xdft[400],
                                           creal_T cfsdft[19200],
                                           real_T b_psidft[19200]);

// Function Definitions
static __global__ __launch_bounds__(64, 1) void cwtMultiAll_kernel1(
    const real_T FourierFactor, const real_T dv1[400], const real_T dv[48],
    real_T b_cf[48], real_T b_psidft[19200])
{
  uint64_T threadId;
  int32_T kk;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  kk = static_cast<int32_T>(threadId);
  if (kk < 48) {
    for (int32_T jj{0}; jj < 400; jj++) {
      if (jj + 1 > 201) {
        b_psidft[kk + 48 * jj] = 0.0;
      } else {
        b_psidft[kk + 48 * jj] =
            2.0 *
            exp(-0.5 * ((dv[kk] * dv1[jj] - 6.0) * (dv[kk] * dv1[jj] - 6.0))) *
            static_cast<real_T>(dv[kk] * dv1[jj] > 0.0);
      }
      if (jj + 1 == 1) {
        b_cf[kk] = FourierFactor / dv[kk];
      }
    }
  }
}

static __global__
    __launch_bounds__(512, 1) void cwtMultiAll_kernel10(creal_T cfsdft[19200])
{
  uint64_T threadId;
  int32_T b_index;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  b_index = static_cast<int32_T>(threadId);
  if (b_index < 19200) {
    real_T ai;
    real_T im;
    real_T re;
    im = cfsdft[b_index].re;
    ai = cfsdft[b_index].im;
    if (ai == 0.0) {
      re = im / 400.0;
      im = 0.0;
    } else if (im == 0.0) {
      re = 0.0;
      im = ai / 400.0;
    } else {
      re = im / 400.0;
      im = ai / 400.0;
    }
    cfsdft[b_index].re = re;
    cfsdft[b_index].im = im;
  }
}

static __global__ __launch_bounds__(512, 1) void cwtMultiAll_kernel11(
    const creal_T cfsdft[19200], creal_T cfs[9600])
{
  uint64_T threadId;
  int32_T b_index;
  int32_T jj;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  jj = static_cast<int32_T>(threadId % 200ULL);
  b_index =
      static_cast<int32_T>((threadId - static_cast<uint64_T>(jj)) / 200ULL);
  if ((static_cast<int32_T>(b_index < 48)) &&
      (static_cast<int32_T>(jj < 200))) {
    cfs[b_index + 48 * jj] = cfsdft[b_index + 48 * (jj + 100)];
  }
}

static __global__ __launch_bounds__(512, 1) void cwtMultiAll_kernel12(
    const creal_T cfs[9600], const int32_T tIndex, creal_T cwtres[96000])
{
  uint64_T threadId;
  int32_T b_index;
  int32_T kk;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  kk = static_cast<int32_T>(threadId % 48ULL);
  b_index =
      static_cast<int32_T>((threadId - static_cast<uint64_T>(kk)) / 48ULL);
  if ((static_cast<int32_T>(b_index < 200)) &&
      (static_cast<int32_T>(kk < 48))) {
    cwtres[(tIndex + 10 * kk) + 480 * b_index] = cfs[kk + 48 * b_index];
  }
}

static __global__
    __launch_bounds__(416, 1) void cwtMultiAll_kernel2(const real_T data[2000],
                                                       real_T xv[400])
{
  uint64_T threadId;
  int32_T kk;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  kk = static_cast<int32_T>(threadId);
  if (kk < 400) {
    if (kk + 1 < 101) {
      xv[kk] = data[99 - kk];
    } else if (kk + 1 <= 300) {
      xv[kk] = data[kk - 100];
    } else {
      xv[kk] = data[499 - kk];
    }
  }
}

static __global__
    __launch_bounds__(224, 1) void cwtMultiAll_kernel3(creal_T xdft[400])
{
  uint64_T threadId;
  int32_T b_index;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  b_index = static_cast<int32_T>(threadId);
  if (b_index < 199) {
    xdft[399 - b_index].re = xdft[b_index + 1].re;
    xdft[399 - b_index].im = -xdft[b_index + 1].im;
  }
}

static __global__ __launch_bounds__(512, 1) void cwtMultiAll_kernel4(
    const creal_T xdft[400], creal_T cfsdft[19200], real_T b_psidft[19200])
{
  uint64_T threadId;
  int32_T jj;
  int32_T kk;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  jj = static_cast<int32_T>(threadId % 400ULL);
  kk = static_cast<int32_T>((threadId - static_cast<uint64_T>(jj)) / 400ULL);
  if ((static_cast<int32_T>(kk < 48)) && (static_cast<int32_T>(jj < 400))) {
    real_T im;
    im = b_psidft[kk + 48 * jj];
    cfsdft[kk + 48 * jj].re = im * xdft[jj].re;
    cfsdft[kk + 48 * jj].im = im * xdft[jj].im;
  }
}

static __global__ __launch_bounds__(224, 1) void cwtMultiAll_kernel5(
    const real_T maxwavcf, const real_T FourierFactor, real_T coi[200])
{
  uint64_T threadId;
  int32_T kk;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  kk = static_cast<int32_T>(threadId);
  if (kk < 200) {
    real_T im;
    int32_T b_kk;
    if (kk + 1 <= 100) {
      b_kk = kk + 1;
    } else if (kk + 1 == 101) {
      b_kk = 100;
    } else {
      b_kk = 200 - kk;
    }
    im =
        1.0 / (0.74048048969306091 * FourierFactor * static_cast<real_T>(b_kk));
    coi[kk] = im;
    if (im > maxwavcf) {
      coi[kk] = maxwavcf;
    }
  }
}

static __global__ __launch_bounds__(64,
                                    1) void cwtMultiAll_kernel6(real_T b_cf[48],
                                                                real_T f[48])
{
  uint64_T threadId;
  int32_T b_index;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  b_index = static_cast<int32_T>(threadId);
  if (b_index < 48) {
    f[b_index] = b_cf[b_index];
  }
}

static __global__ __launch_bounds__(416, 1) void cwtMultiAll_kernel7(
    const real_T data[2000], const int32_T tIndex, real_T xv[400])
{
  uint64_T threadId;
  int32_T kk;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  kk = static_cast<int32_T>(threadId);
  if (kk < 400) {
    if (kk + 1 < 101) {
      xv[kk] = data[(200 * tIndex - kk) + 99];
    } else if (kk + 1 <= 300) {
      xv[kk] = data[(kk + 200 * tIndex) - 100];
    } else {
      xv[kk] = data[(200 * tIndex - kk) + 499];
    }
  }
}

static __global__
    __launch_bounds__(224, 1) void cwtMultiAll_kernel8(creal_T xdft[400])
{
  uint64_T threadId;
  int32_T b_index;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  b_index = static_cast<int32_T>(threadId);
  if (b_index < 199) {
    xdft[399 - b_index].re = xdft[b_index + 1].re;
    xdft[399 - b_index].im = -xdft[b_index + 1].im;
  }
}

static __global__ __launch_bounds__(512, 1) void cwtMultiAll_kernel9(
    const creal_T xdft[400], creal_T cfsdft[19200], real_T b_psidft[19200])
{
  uint64_T threadId;
  int32_T jj;
  int32_T kk;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  jj = static_cast<int32_T>(threadId % 400ULL);
  kk = static_cast<int32_T>((threadId - static_cast<uint64_T>(jj)) / 400ULL);
  if ((static_cast<int32_T>(kk < 48)) && (static_cast<int32_T>(jj < 400))) {
    real_T im;
    im = b_psidft[kk + 48 * jj];
    cfsdft[kk + 48 * jj].re = im * xdft[jj].re;
    cfsdft[kk + 48 * jj].im = im * xdft[jj].im;
  }
}

void cwtMultiAll(const real_T data[2000], real_T fs, creal_T cwtres[96000],
                 real_T f[48], real_T coi[200])
{
  static const real_T dv1[400]{0.0,
                               0.015707963267948967,
                               0.031415926535897934,
                               0.0471238898038469,
                               0.062831853071795868,
                               0.078539816339744828,
                               0.0942477796076938,
                               0.10995574287564278,
                               0.12566370614359174,
                               0.1413716694115407,
                               0.15707963267948966,
                               0.17278759594743864,
                               0.1884955592153876,
                               0.20420352248333656,
                               0.21991148575128555,
                               0.23561944901923451,
                               0.25132741228718347,
                               0.26703537555513246,
                               0.28274333882308139,
                               0.29845130209103038,
                               0.31415926535897931,
                               0.3298672286269283,
                               0.34557519189487729,
                               0.36128315516282622,
                               0.37699111843077521,
                               0.3926990816987242,
                               0.40840704496667313,
                               0.42411500823462212,
                               0.4398229715025711,
                               0.45553093477052004,
                               0.471238898038469,
                               0.48694686130641796,
                               0.50265482457436694,
                               0.51836278784231593,
                               0.53407075111026492,
                               0.5497787143782138,
                               0.56548667764616278,
                               0.58119464091411177,
                               0.59690260418206076,
                               0.61261056745000975,
                               0.62831853071795862,
                               0.64402649398590761,
                               0.6597344572538566,
                               0.67544242052180559,
                               0.69115038378975457,
                               0.70685834705770356,
                               0.72256631032565244,
                               0.73827427359360143,
                               0.75398223686155041,
                               0.7696902001294994,
                               0.78539816339744839,
                               0.80110612666539727,
                               0.81681408993334625,
                               0.83252205320129524,
                               0.84823001646924423,
                               0.86393797973719322,
                               0.87964594300514221,
                               0.89535390627309108,
                               0.91106186954104007,
                               0.92676983280898906,
                               0.942477796076938,
                               0.958185759344887,
                               0.97389372261283591,
                               0.9896016858807849,
                               1.0053096491487339,
                               1.0210176124166828,
                               1.0367255756846319,
                               1.0524335389525807,
                               1.0681415022205298,
                               1.0838494654884787,
                               1.0995574287564276,
                               1.1152653920243767,
                               1.1309733552923256,
                               1.1466813185602747,
                               1.1623892818282235,
                               1.1780972450961724,
                               1.1938052083641215,
                               1.2095131716320704,
                               1.2252211349000195,
                               1.2409290981679684,
                               1.2566370614359172,
                               1.2723450247038663,
                               1.2880529879718152,
                               1.3037609512397643,
                               1.3194689145077132,
                               1.3351768777756623,
                               1.3508848410436112,
                               1.36659280431156,
                               1.3823007675795091,
                               1.398008730847458,
                               1.4137166941154071,
                               1.429424657383356,
                               1.4451326206513049,
                               1.460840583919254,
                               1.4765485471872029,
                               1.492256510455152,
                               1.5079644737231008,
                               1.5236724369910497,
                               1.5393804002589988,
                               1.5550883635269477,
                               1.5707963267948968,
                               1.5865042900628457,
                               1.6022122533307945,
                               1.6179202165987436,
                               1.6336281798666925,
                               1.6493361431346416,
                               1.6650441064025905,
                               1.6807520696705394,
                               1.6964600329384885,
                               1.7121679962064373,
                               1.7278759594743864,
                               1.7435839227423353,
                               1.7592918860102844,
                               1.7749998492782333,
                               1.7907078125461822,
                               1.8064157758141313,
                               1.8221237390820801,
                               1.8378317023500292,
                               1.8535396656179781,
                               1.869247628885927,
                               1.8849555921538761,
                               1.900663555421825,
                               1.9163715186897741,
                               1.9320794819577229,
                               1.9477874452256718,
                               1.9634954084936209,
                               1.9792033717615698,
                               1.9949113350295189,
                               2.0106192982974678,
                               2.0263272615654166,
                               2.0420352248333655,
                               2.0577431881013148,
                               2.0734511513692637,
                               2.0891591146372126,
                               2.1048670779051615,
                               2.1205750411731104,
                               2.1362830044410597,
                               2.1519909677090086,
                               2.1676989309769574,
                               2.1834068942449063,
                               2.1991148575128552,
                               2.2148228207808045,
                               2.2305307840487534,
                               2.2462387473167023,
                               2.2619467105846511,
                               2.2776546738526,
                               2.2933626371205493,
                               2.3090706003884982,
                               2.3247785636564471,
                               2.340486526924396,
                               2.3561944901923448,
                               2.3719024534602942,
                               2.387610416728243,
                               2.4033183799961919,
                               2.4190263432641408,
                               2.4347343065320897,
                               2.450442269800039,
                               2.4661502330679879,
                               2.4818581963359367,
                               2.4975661596038856,
                               2.5132741228718345,
                               2.5289820861397838,
                               2.5446900494077327,
                               2.5603980126756816,
                               2.5761059759436304,
                               2.5918139392115798,
                               2.6075219024795286,
                               2.6232298657474775,
                               2.6389378290154264,
                               2.6546457922833753,
                               2.6703537555513246,
                               2.6860617188192735,
                               2.7017696820872223,
                               2.7174776453551712,
                               2.73318560862312,
                               2.7488935718910694,
                               2.7646015351590183,
                               2.7803094984269672,
                               2.7960174616949161,
                               2.8117254249628649,
                               2.8274333882308142,
                               2.8431413514987631,
                               2.858849314766712,
                               2.8745572780346609,
                               2.8902652413026098,
                               2.9059732045705591,
                               2.921681167838508,
                               2.9373891311064568,
                               2.9530970943744057,
                               2.9688050576423546,
                               2.9845130209103039,
                               3.0002209841782528,
                               3.0159289474462017,
                               3.0316369107141505,
                               3.0473448739820994,
                               3.0630528372500487,
                               3.0787608005179976,
                               3.0944687637859465,
                               3.1101767270538954,
                               3.1258846903218442,
                               3.1415926535897936,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0,
                               0.0};
  static const real_T dv[48]{
      2.5929415186851879, 2.7790459096353382, 2.9785076571171887,
      3.1922854648593515, 3.4214068460765876, 3.6669730621648822,
      3.9301644158637692, 4.21224592432724,   4.5145733993705948,
      4.838599964117666,  5.1858830373703739, 5.5580918192706745,
      5.9570153142343765, 6.3845709297187012, 6.8428136921531726,
      7.3339461243297626, 7.8603288317275366, 8.424491848654478,
      9.029146798741186,  9.6771999282353285, 10.371766074740744,
      11.116183638541346, 11.914030628468748, 12.769141859437399,
      13.685627384306342, 14.66789224865952,  15.720657663455068,
      16.848983697308949, 18.058293597482365, 19.35439985647065,
      20.743532149481485, 22.232367277082687, 23.828061256937492,
      25.538283718874787, 27.371254768612673, 29.335784497319032,
      31.441315326910129, 33.697967394617891, 36.116587194964723,
      38.708799712941286, 41.487064298962956, 44.464734554165354,
      47.656122513874962, 51.076567437749567, 54.742509537225331,
      58.671568994638044, 62.882630653820236, 67.395934789235753};
  hipfftHandle b_fftPlanHandle;
  hipfftHandle fftPlanHandle;
  creal_T(*gpu_cfsdft)[19200];
  creal_T(*gpu_cfs)[9600];
  creal_T(*gpu_xdft)[400];
  real_T(*gpu_psidft)[19200];
  real_T(*gpu_xv)[400];
  real_T(*gpu_cf)[48];
  real_T maxwavcf;
  int32_T inembed;
  if (!gpuConstsCopied_cwtMultiAll) {
    gpuConstsCopied_cwtMultiAll = true;
    hipMemcpy(*dv1_gpu_clone, dv1, sizeof(real_T[400]),
               hipMemcpyHostToDevice);
    hipMemcpy(*dv_gpu_clone, dv, sizeof(real_T[48]), hipMemcpyHostToDevice);
  }
  hipMalloc(&gpu_cfs, 153600ULL);
  hipMalloc(&gpu_cfsdft, 307200ULL);
  hipMalloc(&gpu_xdft, 6400ULL);
  hipMalloc(&gpu_psidft, 153600ULL);
  hipMalloc(&gpu_cf, 384ULL);
  hipMalloc(&gpu_xv, 3200ULL);
  //  Apply cwt to multi-channel data. The result is returned in a
  //  nTrial*nFreq*nTime complex double matrix. This procedure is for
  //  cross-spectral density matrix computation in nonparametric computation of
  //  granger causality.
  //
  //  It can be encoded by gpucoder for parallel computation. See mGpucoder.m
  if (!psidft_not_empty) {
    hipMemcpy(*gpu_cf, cf, 384ULL, hipMemcpyHostToDevice);
    hipMemcpy(*gpu_psidft, psidft, 153600ULL, hipMemcpyHostToDevice);
    cwtMultiAll_kernel1<<<dim3(1U, 1U, 1U), dim3(64U, 1U, 1U)>>>(
        0.954929658551372 * fs, *dv1_gpu_clone, *dv_gpu_clone, *gpu_cf,
        *gpu_psidft);
    hipMemcpy(psidft, *gpu_psidft, 153600ULL, hipMemcpyDeviceToHost);
    hipMemcpy(cf, *gpu_cf, 384ULL, hipMemcpyDeviceToHost);
    psidft_not_empty = true;
  }
  cwtMultiAll_kernel2<<<dim3(1U, 1U, 1U), dim3(416U, 1U, 1U)>>>(data, *gpu_xv);
  inembed = 400;
  fftPlanHandle = acquireCUFFTPlan(1, &inembed, &inembed, 1, 1, HIPFFT_D2Z, 1);
  hipfftExecD2Z(fftPlanHandle, (hipfftDoubleReal *)&(*gpu_xv)[0],
               (hipfftDoubleComplex *)&(*gpu_xdft)[0]);
  cwtMultiAll_kernel3<<<dim3(1U, 1U, 1U), dim3(224U, 1U, 1U)>>>(*gpu_xdft);
  hipMemcpy(*gpu_psidft, psidft, 153600ULL, hipMemcpyHostToDevice);
  cwtMultiAll_kernel4<<<dim3(38U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
      *gpu_xdft, *gpu_cfsdft, *gpu_psidft);
  hipMemcpy(psidft, *gpu_psidft, 153600ULL, hipMemcpyDeviceToHost);
  inembed = 400;
  b_fftPlanHandle =
      acquireCUFFTPlan(1, &inembed, &inembed, 48, 1, HIPFFT_Z2Z, 48);
  hipfftExecZ2Z(b_fftPlanHandle, (hipfftDoubleComplex *)&(*gpu_cfsdft)[0],
               (hipfftDoubleComplex *)&(*gpu_cfsdft)[0], HIPFFT_BACKWARD);
  maxwavcf = cf[0];
  for (inembed = 0; inembed < 47; inembed++) {
    boolean_T p;
    if (std::isnan(cf[inembed + 1])) {
      p = false;
    } else if (std::isnan(maxwavcf)) {
      p = true;
    } else {
      p = (maxwavcf < cf[inembed + 1]);
    }
    if (p) {
      maxwavcf = cf[inembed + 1];
    }
  }
  cwtMultiAll_kernel5<<<dim3(1U, 1U, 1U), dim3(224U, 1U, 1U)>>>(maxwavcf,
                                                                1.0 / fs, coi);
  hipMemcpy(*gpu_cf, cf, 384ULL, hipMemcpyHostToDevice);
  cwtMultiAll_kernel6<<<dim3(1U, 1U, 1U), dim3(64U, 1U, 1U)>>>(*gpu_cf, f);
  hipMemcpy(cf, *gpu_cf, 384ULL, hipMemcpyDeviceToHost);
  for (int32_T tIndex{0}; tIndex < 10; tIndex++) {
    hipfftHandle c_fftPlanHandle;
    hipfftHandle d_fftPlanHandle;
    cwtMultiAll_kernel7<<<dim3(1U, 1U, 1U), dim3(416U, 1U, 1U)>>>(data, tIndex,
                                                                  *gpu_xv);
    inembed = 400;
    c_fftPlanHandle =
        acquireCUFFTPlan(1, &inembed, &inembed, 1, 1, HIPFFT_D2Z, 1);
    hipfftExecD2Z(c_fftPlanHandle, (hipfftDoubleReal *)&(*gpu_xv)[0],
                 (hipfftDoubleComplex *)&(*gpu_xdft)[0]);
    cwtMultiAll_kernel8<<<dim3(1U, 1U, 1U), dim3(224U, 1U, 1U)>>>(*gpu_xdft);
    hipMemcpy(*gpu_psidft, psidft, 153600ULL, hipMemcpyHostToDevice);
    cwtMultiAll_kernel9<<<dim3(38U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
        *gpu_xdft, *gpu_cfsdft, *gpu_psidft);
    hipMemcpy(psidft, *gpu_psidft, 153600ULL, hipMemcpyDeviceToHost);
    inembed = 400;
    d_fftPlanHandle =
        acquireCUFFTPlan(1, &inembed, &inembed, 48, 1, HIPFFT_Z2Z, 48);
    hipfftExecZ2Z(d_fftPlanHandle, (hipfftDoubleComplex *)&(*gpu_cfsdft)[0],
                 (hipfftDoubleComplex *)&(*gpu_cfsdft)[0], HIPFFT_BACKWARD);
    cwtMultiAll_kernel10<<<dim3(38U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
        *gpu_cfsdft);
    cwtMultiAll_kernel11<<<dim3(19U, 1U, 1U), dim3(512U, 1U, 1U)>>>(*gpu_cfsdft,
                                                                    *gpu_cfs);
    cwtMultiAll_kernel12<<<dim3(19U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
        *gpu_cfs, tIndex, cwtres);
  }
  hipFree(*gpu_xv);
  hipFree(*gpu_cf);
  hipFree(*gpu_psidft);
  hipFree(*gpu_xdft);
  hipFree(*gpu_cfsdft);
  hipFree(*gpu_cfs);
}

// End of code generation (cwtMultiAll.cu)
