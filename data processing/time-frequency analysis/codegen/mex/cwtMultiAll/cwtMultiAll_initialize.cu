//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// cwtMultiAll_initialize.cu
//
// Code generation for function 'cwtMultiAll_initialize'
//

// Include files
#include "cwtMultiAll_initialize.h"
#include "_coder_cwtMultiAll_mex.h"
#include "cwt.h"
#include "cwtMultiAll_data.h"
#include "rt_nonfinite.h"

// Function Declarations
static void cwtMultiAll_once();

// Function Definitions
static void cwtMultiAll_once()
{
  mex_InitInfAndNan();
  psidft_not_empty_init();
  hipMalloc(&dv1_gpu_clone, sizeof(real_T[400]));
  hipMalloc(&dv_gpu_clone, sizeof(real_T[48]));
}

void cwtMultiAll_initialize()
{
  mexFunctionCreateRootTLS();
  emlrtClearAllocCountR2012b(emlrtRootTLSGlobal, false, 0U, nullptr);
  emlrtEnterRtStackR2012b(emlrtRootTLSGlobal);
  emlrtLicenseCheckR2012b(emlrtRootTLSGlobal,
                          (const char_T *)"distrib_computing_toolbox", 2);
  emlrtLicenseCheckR2012b(emlrtRootTLSGlobal, (const char_T *)"wavelet_toolbox",
                          2);
  if (emlrtFirstTimeR2012b(emlrtRootTLSGlobal)) {
    cwtMultiAll_once();
  }
  emlrtInitGPU(emlrtRootTLSGlobal);
  hipGetLastError();
}

// End of code generation (cwtMultiAll_initialize.cu)
