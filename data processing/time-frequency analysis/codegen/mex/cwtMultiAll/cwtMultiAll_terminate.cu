//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// cwtMultiAll_terminate.cu
//
// Code generation for function 'cwtMultiAll_terminate'
//

// Include files
#include "cwtMultiAll_terminate.h"
#include "_coder_cwtMultiAll_mex.h"
#include "cwtMultiAll_data.h"
#include "rt_nonfinite.h"

// Function Definitions
void cwtMultiAll_atexit()
{
  mexFunctionCreateRootTLS();
  emlrtEnterRtStackR2012b(emlrtRootTLSGlobal);
  emlrtLeaveRtStackR2012b(emlrtRootTLSGlobal);
  emlrtDestroyRootTLS(&emlrtRootTLSGlobal);
  emlrtExitTimeCleanup(&emlrtContextGlobal);
  hipFree(*dv1_gpu_clone);
  hipFree(*dv_gpu_clone);
}

void cwtMultiAll_terminate()
{
  hipError_t errCode;
  errCode = hipGetLastError();
  if (errCode != hipSuccess) {
    emlrtThinCUDAError(static_cast<uint32_T>(errCode),
                       (char_T *)hipGetErrorName(errCode),
                       (char_T *)hipGetErrorString(errCode),
                       (char_T *)"SafeBuild", emlrtRootTLSGlobal);
  }
  emlrtLeaveRtStackR2012b(emlrtRootTLSGlobal);
  emlrtDestroyRootTLS(&emlrtRootTLSGlobal);
}

// End of code generation (cwtMultiAll_terminate.cu)
